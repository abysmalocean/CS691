#include "hip/hip_runtime.h"
/*
   Liang Xu 2016
 */
 #define PI (3.141592653589793238462643383279)
#define E  (2.718281828459045235360287471352)
#define L(i) ((int64_t)i)
#define D(i) ((double)i)

#include <string>
#include <iostream>
#include <cstring>
#include <cstdio>
#include <fstream>
#include <sstream>
#include <sys/time.h>
#include <cstdio>
#include <unistd.h>
#include <stdio.h>
#include <sstream>
#include <vector>
#include <fstream>
#include <string>
#include <cstring>
#include <cstdlib>
#include <cmath>
#include <ctime>
int dimension = 1000000;
using namespace std;

__global__ void matrixAdd(float *A, float *B, float *C, int numElements)
{


}

double** readOvectorVec()
{
  int dimension_fileread = 1000;
  double* d = new double[dimension];
  stringstream ss;
  ss<< "cdatafiles/" << "F3" << "-xopt.txt";
  ifstream file (ss.str());
  string value;
  string line;
  int c=0;

  if (file.is_open())
    {
      stringstream iss;
      while ( getline(file, line) )
        {
          iss<<line;
          while (getline(iss, value, ','))
            {
              d[c++] = stod(value);
            }
          iss.clear();
          // if (c==dimension)
          //   {
          //     break;
          //   }
          // printf("%d\n",c);
        }
      file.close();
    }
  else
    {
      cout<<"Cannot open datafiles"<<endl;
    }
  int tid = 0;
  for (int i = 1; i < 1000; i++) {
    tid = i * 1000;
    for (int j = 0; j < dimension_fileread; j++) {
      d[tid+j] = d[j];
      //printf("d[%d] = d[%d] ---> %f\n",tid+j,j,d[tid+j] );
    }
  }
  return d;
}

AckleyCompute(double*x)
{
        int i;
        double result;
        anotherz = new double[dimension];
        Ovector = NULL;
        minX = -32;
        maxX = 32;
        ID = 3;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;

        if(Ovector == NULL) {
                // Ovector = createShiftVector(dimension,minX,maxX);
                Ovector = readOvector();
        }
/*
        for(i = dimension - 1; i >= 0; i--) {
                anotherz[i] = x[i] - Ovector[i];
                //printf("%d\n",dimension );
        }
        hipEventRecord(start);
        result = ackley(anotherz,dimension);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("CPU running time (Only the calculation not include the inilization) %f ms\n", milliseconds);
*/
        return(result);
}

int main(int argc, char* argv[])
{
        double* X;
        double resultTest;
        unsigned dim = 1000000;
        unsigned run = 1;
        char *p;
        if (argc == 2)
        {
                run = (unsigned)strtol(argv[1], &p, 10);
        }
        X = new double[dim];
        for (unsigned i=0; i<dim; i++) {
                X[i]=0;
        }
        printf("Ackley Function value = %1.20E\n", resultTest = AckleyCompute(X));

/*
    gettimeofday(&start, NULL);
    for (unsigned j=0; j < run; j++){
      if (fp->compute(X) - resultTest > 0.1)
      {
        printf("result not equal to the previous result\n" );
      }
    }
    gettimeofday(&end, NULL);

    seconds  = end.tv_sec  - start.tv_sec;
    useconds = end.tv_usec - start.tv_usec;

    mtime = (((seconds) * 1000 + useconds/1000.0) + 0.5)/1000;

    runTimeVec.push_back(mtime);
    //printf ( "F %d, Running Time = %f s\n\n", fp->getID(), mtime);

    //TODO GPU Computing
    fp->GPUcompute(X,resultTest,run);

    delete fp;

   delete []X;

   // for (unsigned i=0; i<runTimeVec.size(); i++){
   //   printf ( "%f\n", runTimeVec[i] );
   // }
 */
        return 0;
        return 0;
}
