#include "hip/hip_runtime.h"
#include "Benchmarks.h"
extern "C"
double GPUcompute(double* x,double resultTest, unsigned run);

/*******************************************************************************
   //************Cuda kernel Function for Midterm  test****************************
   //************Author Liang Xu***************************************************
   //************Dtate 10.26 2016**************************************************
   //******************************************************************************
 ********************************************************************************/

__global__ void ackleyKernel(double* z, int dim,double * d_sum1, double * d_sum2,int stream, int numberElementsPerStream ){
        //printf("Liang XU in kernel\n");
        /*******************************************************************************
           //************ First need to calcuate the X[i]**********************************
         ********************************************************************************/

        int tid = ( blockDim.x * blockIdx.x ) + threadIdx.x;
        __shared__ double sharedSum1Array[512]; // size of the share memory is the size of block
        __shared__ double sharedSum2Array[512]; // size of the share memory is the size of block
        dim = numberElementsPerStream;
        double hat_x;
        double c1;
        double c2;
        int sign;
        double beta = 0.2;
        double alpha = 10;
        double x;
        double sum1;
        double sum2;

        x = z[tid];
        //x =1;
        if (x == 0) {
                hat_x = 0;
        }else{hat_x = log(abs(x)); }

        if (x>0)
        {
                c1 = 10;
                c2 = 7.9;
                sign = 1;
        }
        else
        {
                c1 = 5.5;
                c2 = 3.1;
                if(x != 0)
                {
                        sign  = -1;
                }else{sign = 0; }
        }
        x = sign * exp( hat_x + 0.049 * ( sin( c1 * hat_x ) + sin( c2* hat_x )  ) );

        if(x > 0)
        {
                x = pow(x, 1 + beta * tid/((double) (dim-1)) * sqrt(x) );
        }
        x = x * pow(alpha, 0.5 * tid/((double) (dim-1)) );
        sum1 = x * x;
        sum2 = cos(2.0 * PI * x);
        //printf("block dim is [%d]threadID is [%d] sum1 is [%f] sum2 is [%f]\n",blockDim.x,tid,sum1,sum2 );
        //printf("block dim is [%d] threadIdx[%d]\n",blockDim.x,threadIdx.x );
        /*******************************************************************************
           //************Secod step is use reduction method calculate the sum**************
         ********************************************************************************/
        // First setp to save the sum1 and sum2 local data to the shard memory

        sharedSum1Array[threadIdx.x] = (tid < dim) ? sum1 : 0;
        sharedSum2Array[threadIdx.x] = (tid < dim) ? sum2 : 0;
        __syncthreads();
        //printf("Liang\n" );
        //Step 2, reduction
        for (int s = (blockDim.x)/2; s > 0; s >>= 1) {
                //printf("s = [%d]\n",s );
                //printf("threadIdx.x  = [%d]\n",threadIdx.x  );

                if(threadIdx.x < s)
                {
                        //printf("threadIdx.x + s = [%d]\n",threadIdx.x + s );
                        sharedSum1Array[threadIdx.x] += sharedSum1Array[threadIdx.x + s];
                        sharedSum2Array[threadIdx.x] += sharedSum2Array[threadIdx.x + s];
                }
                //printf("Liang\n" );
                __syncthreads();
        }
        //step 3, annd to the final output
        if(threadIdx.x == 0)
        {
                //atomicAdd((float *)d_sum1,(float)sharedSum1Array[0]);
                //atomicAdd((float *)d_sum2,(float)sharedSum2Array[0]);
                // for Cuda 8.0 it support double atomic add
                //atomicAdd(d_sum1,sharedSum1Array[0]);
                //atomicAdd(d_sum2,sharedSum2Array[0]);
                //printf(" sum1[%f]--> [%f] sum2 [%f]-->[%f]\n", *d_sum1,*d_sum1-sharedSum1Array[0],*d_sum2 ,*d_sum2-sharedSum2Array[0]);
                d_sum1[blockIdx.x] = sharedSum1Array[0];
                d_sum2[blockIdx.x] = sharedSum2Array[0];
        }
}



void cudaErrorCheck(hipError_t hipError_t)
{
        if(hipError_t != hipSuccess)
        {
                fprintf(stderr, "cudaGetError returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
                //exit(EXIT_FAILURE);
        }else
        {
                printf("--> No error PASS\n");
        }

}


//TODO GPU computing
//fp->GPUcompute(X,resultTest,run);
double GPUcompute(double* anotherz,double resultTest, unsigned run){

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;
        hipError_t hipError_t;
        double sum1;
        double sum2;
        double result = 0;
        int dimension = 1000000;

        int numStreams = 1;
        hipStream_t *streams = (hipStream_t*) malloc (numStreams * sizeof(hipStream_t));
        for (int i = 0; i < numStreams; i++)
                hipStreamCreate(&streams[i]);

        //printf("\n*****************GPU Computing Result***************************\n\n");

        // inilize the CUDA
        int threadsPerBlockDim = 512;
        int numberElementsPerStream = (dimension + numStreams - 1) / numStreams;
        int gridDimSize_orig = (dimension + threadsPerBlockDim - 1) / threadsPerBlockDim;
        int gridDimSize = (numberElementsPerStream + threadsPerBlockDim - 1) / threadsPerBlockDim;

        dim3 blockSize(threadsPerBlockDim);
        dim3 gridSize (gridDimSize);


        //printf("GPU is running on Blocksize[%d],gridSize[%d]\n",threadsPerBlockDim,gridDimSize );
        // allocate the memory to device
        double *d_x;
        double *d_sum1, *d_sum2;
        double *h_sum1, *h_sum2;
        h_sum1 = (double *)malloc(gridDimSize_orig * sizeof(double));
        h_sum2 = (double *)malloc(gridDimSize_orig * sizeof(double));

        hipError_t =  hipMalloc(&d_x,dimension * sizeof(double));
        //printf("hipMalloc(&d_x,dimension * sizeof(double));                    "); cudaErrorCheck(hipError_t);
        hipError_t =  hipMalloc(&d_sum1,gridDimSize_orig*sizeof(double));
        //printf("hipMalloc(&d_sum1,sizeof(double));                             "); cudaErrorCheck(hipError_t);
        hipError_t =  hipMalloc(&d_sum2,gridDimSize_orig*sizeof(double));
        //printf("hipMalloc(&d_sum2,sizeof(double));                             "); cudaErrorCheck(hipError_t);


        //Event start
        hipEventRecord(start);
        for (int i = 0; i < numStreams; i++)
        {

                hipError_t =  hipMemcpyAsync(&d_x[i*numberElementsPerStream], &anotherz[i*numberElementsPerStream], numberElementsPerStream * sizeof(double), hipMemcpyHostToDevice,streams[i]);
                //printf(" hipMemcpy(d_x, anotherz, dimension * sizeof(double), hipMemcpyHostToDevice);"); cudaErrorCheck(hipError_t);
                //hipError_t =  hipMemset(d_sum1, 0.000, sizeof(float));
                //printf("hipMemset(d_sum1, 0.000, sizeof(double));"); cudaErrorCheck(hipError_t);
                //hipError_t =  hipMemset(d_sum2, 0.000, sizeof(float));
                //printf("hipMemset(d_sum2, 0.000, sizeof(double));"); cudaErrorCheck(hipError_t);

                ackleyKernel<<<gridSize, blockSize, 0, streams[i]>>>(d_x,dimension,d_sum1,d_sum2,i,numberElementsPerStream);

                //result = ackley(anotherz,dimension);

                hipError_t = hipGetLastError();
                //printf("Kernek Function                                                 ");cudaErrorCheck(hipError_t);
                hipError_t = hipMemcpyAsync(&h_sum1[i*gridDimSize], d_sum1, gridDimSize*sizeof(double), hipMemcpyDeviceToHost,streams[i]);
                //printf("hipMemcpy(&sum1, d_sum1, sizeof(float), hipMemcpyDeviceToHost)"); cudaErrorCheck(hipError_t);
                hipError_t = hipMemcpyAsync(&h_sum2[i*gridDimSize], d_sum2, gridDimSize*sizeof(double), hipMemcpyDeviceToHost,streams[i]);
                //printf("hipMemcpy(&sum2, d_sum2, sizeof(float), hipMemcpyDeviceToHost)"); cudaErrorCheck(hipError_t);
                //printf(" cpoy from kernel is %f\n",sum1 );
        }

        for(int i = 0; i < gridDimSize; i++)
        {
                sum1 = sum1 + h_sum1[i];
                sum2 = sum2 + h_sum2[i];
        }

        result = -20.0 * exp(-0.2 * sqrt(sum1/dimension)) - exp(sum2/dimension) + 20.0 + E;

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        // Testing the result
        if (abs(result - resultTest) > 0.00001)
        {
                printf("Result in GPU is %f\n",result );
                printf("result not equal to the previous result in GPU computing\n" );
        }else{printf("GUP return the correct result\n"); }

        printf("GUP result is = %1.20E\n", result );
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("========================================> GPU running time is %f ms\n", milliseconds);

        hipFree(d_x);
        hipFree(d_sum1);
        hipFree(d_sum2);
        free(h_sum1);
        free(h_sum2);
        return(result);
}
