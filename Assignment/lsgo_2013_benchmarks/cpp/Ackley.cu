#include "hip/hip_runtime.h"
#include "Benchmarks.h"

extern "C"
double GPUcompute(double* x,double resultTest, unsigned run);

/*******************************************************************************
   //************Cuda kernel Function for Midterm  test****************************
   //************Author Liang Xu***************************************************
   //************Dtate 10.26 2016**************************************************
   //******************************************************************************
 ********************************************************************************/
 __global__ void ackley_kernel(double *d_x, double *sum1, double *sum2, int dim){
  //int BLOCK_SIZE = 512;
 	__shared__ double sm[512];
 	__shared__ double sm_cos[512];

 	int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
 	int tid = threadIdx.x;

 	sm[tid] = d_x[global_tid];

 	// initialization of sign, hat, c1, c2
 	int sign;
 	if( sm[tid] == 0 )
 		sign = 0;
 	else
 		sign = sm[tid]>0 ? 1:-1;

 	double hat;
 	if(sm[tid] == 0)
 		hat = 0;
 	else
 		hat = log(abs(sm[tid]));

 	double c1;
 	if(sm[tid]>0)
 		c1 = 10;
 	else
 		c1 = 5.5;

 	double c2;
 	if( sm[tid]>0 )
 		c2 = 7.9;
 	else
 		c2=3.1;

 	// transform osz
 	sm[tid] = sign * exp(hat + 0.049 * (sin(c1*hat) + sin(c2*hat)));

 	// transform asy
 	if(sm[tid]>0)
 		sm[tid] = pow(sm[tid], 1+0.2* global_tid/(double)(dim-1) * sqrt(sm[tid]));

 	// lambda
 	sm[tid] = sm[tid] * pow( 10.0, 0.5* global_tid/((double)(dim-1)) );

 	// cos(2.0 * pi * x[i])
 	sm_cos[tid] = cos(2.0 * PI * sm[tid]);

 	// x square
 	sm[tid] = sm[tid]*sm[tid];

 	__syncthreads();

 	// reduction
 	for( int i=512/2; i>0; i>>=1 ){
 		if(tid<i){
 			sm[tid] += sm[tid+i];
 			sm_cos[tid] += sm_cos[tid+i];
 		}
 	}
 	__syncthreads();

 	// get the value from first element of shared memory
 	if(tid == 0){
 		sum1[blockIdx.x] = sm[tid];
 		sum2[blockIdx.x] = sm_cos[tid];
 	}
 }

__global__ void ackleyKernel(double* z, int dim,double* d_sum1, double* d_sum2){
  printf("Liang XU in kernel\n");
/*******************************************************************************
   //************ First need to calcuate the X[i]**********************************
 ********************************************************************************/

    int tid = ( blockDim.x * blockIdx.x ) + threadIdx.x;
    __shared__ double sharedSum1Array[1024]; // size of the share memory is the size of block
    __shared__ double sharedSum2Array[1024]; // size of the share memory is the size of block
/*
    double hat_x;
    double c1;
    double c2;
    int sign;
    double beta = 0.2;
    double alpha = 10;
    double x;
    double sum1;
    double sum2;

    x = z[tid];
    if (x == 0) {
      hat_x = 0;
    }else{hat_x = log(abs(x));}

    if (x>0)
    {
        c1 = 10;
        c2 = 7.9;
        sign = 1;
    }
    else
    {
      c1 = 5.5;
      c2 = 3.1;
      if(x != 0)
      {
        sign  = -1 ;
      }else{sign = 0;}
    }
    x = sign * exp( hat_x + 0.049 * ( sin( c1 * hat_x ) + sin( c2* hat_x )  ) ) ;

    if(x > 0)
    {
      x = pow(x, 1 + beta * tid/((double) (dim-1)) * sqrt(x) );
    }
    x = x * pow(alpha, 0.5 * tid/((double) (dim-1)) );
    sum1 = x * x;
    sum2 = cos(2.0 * PI * x);
 */
        /*******************************************************************************
           //************Secod step is use reduction method calculate the sum**************
         ********************************************************************************/
        // First setp to save the sum1 and sum2 local data to the shard memory
        /*
           sharedSum1Array[threadIdx.x] = (tid < dim) ? sum1 : 0;
           sharedSum2Array[threadIdx.x] = (tid < dim) ? sum2 : 0;
           __syncthreads();
           //Step 2, reduction
           for (int s = blockDim.x/2; s > 0 ; s >>= 1) {
           if(threadIdx.x < s)
            sharedSum1Array[threadIdx.x] += sharedSum1Array[threadIdx.x + s];
            sharedSum2Array[threadIdx.x] += sharedSum2Array[threadIdx.x + s];
            __syncthreads();
           }
           //step 3, annd to the final output
           if(threadIdx.x == 0)
           {
           //atomicAdd((float *)d_sum1,(float)sharedSum1Array[0]);
           //atomicAdd((float *)d_sum2,(float)sharedSum2Array[0]);
           // for Cuda 8.0 it support double atomic add
           atomicAdd(d_sum1,sharedSum1Array[0]);
           atomicAdd(d_sum2,sharedSum2Array[0]);
           }


         */
}

void cudaErrorCheck(hipError_t hipError_t)
{
        if(hipError_t != hipSuccess)
        {
                fprintf(stderr, "cudaGetError returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
                //exit(EXIT_FAILURE);
        }else
        {
                printf("--> PASS\n");
        }

}


//TODO GPU computing
//fp->GPUcompute(X,resultTest,run);
double GPUcompute(double* anotherz,double resultTest, unsigned run){

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        float milliseconds = 0;
        hipError_t hipError_t;
        double sum1;
        double sum2;
        double result = 0;
        int dimension = 1000000;

        printf("\n*****************GPU Computing Result***************************\n\n");

        // inilize the CUDA
        int threadsPerBlockDim = 512;
        int gridDimSize = (dimension + threadsPerBlockDim - 1) / threadsPerBlockDim;

        dim3 blockSize(threadsPerBlockDim);
        dim3 gridSize (gridDimSize);


        printf("GPU is running on Blocksize[%d],gridSize[%d]\n",threadsPerBlockDim,gridDimSize );
        // allocate the memory to device
        double *d_x, *d_sum1, *d_sum2;

        hipError_t =  hipMalloc(&d_x,dimension * sizeof(double));
        printf("hipMalloc(&d_x,dimension * sizeof(double));"); cudaErrorCheck(hipError_t);
        hipError_t =  hipMalloc(&d_sum1,sizeof(double));
        printf("hipMalloc(&d_sum1,sizeof(double));"); cudaErrorCheck(hipError_t);
        hipError_t =  hipMalloc(&d_sum2,sizeof(double));
        printf("hipMalloc(&d_sum2,sizeof(double));"); cudaErrorCheck(hipError_t);

        //Event start
        hipEventRecord(start);
        hipError_t =  hipMemcpy(d_x, anotherz, dimension * sizeof(double), hipMemcpyHostToDevice);
        printf(" hipMemcpy(d_x, anotherz, dimension * sizeof(double), hipMemcpyHostToDevice);"); cudaErrorCheck(hipError_t);
        hipError_t =  hipMemset(d_sum1, 0.000, sizeof(double));
        printf("hipMemset(d_sum1, 0.000, sizeof(double));"); cudaErrorCheck(hipError_t);
        hipError_t =  hipMemset(d_sum2, 0.000, sizeof(double));
        printf("hipMemset(d_sum2, 0.000, sizeof(double));"); cudaErrorCheck(hipError_t);

        //ackleyKernel<<<gridSize, blockSize>>>(d_x,dimension,d_sum1,d_sum2);
        //ackleyKernel<<<gridSize, blockSize>>>();

        hipError_t = hipGetLastError();cudaErrorCheck(hipError_t);
        ackley_kernel<<< gridSize, blockSize >>>(d_x, d_sum1, d_sum2, dimension);
        hipError_t = hipGetLastError();cudaErrorCheck(hipError_t);
        //result = ackley(anotherz,dimension);
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        //printf("Liang Xu\n" );
        hipMemcpy(&sum1, d_sum1, sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(&sum2, d_sum2, sizeof(double), hipMemcpyDeviceToHost);


        result = sum1 + sum2;
        printf("Liang Xu\n" );
        if (abs(result - resultTest) > 0.1)
        {
                printf("Result in GPU is %f\n",result );
                printf("result not equal to the previous result in GPU computing\n" );
        }else{printf("GUP return the correct result\n"); }
        hipEventElapsedTime(&milliseconds, start, stop);
        printf("GPU running time is %f ms\n", milliseconds);
        return(result);
}
