#include "hip/hip_runtime.h"
#include "F3.h"

/**
 * Shifted Ackley's Function
 *
 * as defined in "Benchmark Functions for the CEC'2010 Special Session
 * and Competition on Large-Scale Global Optimization" by Ke Tang,
 * Xiaodong Li, P. N. Suganthan, Zhenyu Yang, and Thomas Weise
 * published as technical report on January 8, 2010 at Nature Inspired
 * Computation and Applications Laboratory (NICAL), School of Computer
 * Science and Technology, University of Science and Technology of China,
 * Hefei, Anhui, China.
 */

F3::F3():Benchmarks(){
  Ovector = NULL;
  minX = -32;
  maxX = 32;
  ID = 3;
  anotherz = new double[dimension];

}

F3::~F3(){
  delete[] Ovector;
  delete[] anotherz;
}

/*******************************************************************************
//************Cuda kernel Function for Midterm  test****************************
//************Author Liang Xu***************************************************
//************Dtate 10.26 2016**************************************************
//******************************************************************************
********************************************************************************/
__global__ void testKernel(double* z, int dim,double* d_sum1, double* d_sum2)
{
  printf("Liang XU\n");
/*******************************************************************************
//************ First need to calcuate the X[i]**********************************
********************************************************************************/
/*
    int tid = ( blockDim.x * blockIdx.x ) + threadIdx.x;
    __shared__ double sharedSum1Array[1024]; // size of the share memory is the size of block
    __shared__ double sharedSum2Array[1024]; // size of the share memory is the size of block
    double hat_x;
    double c1;
    double c2;
    int sign;
    double beta = 0.2;
    double alpha = 10;
    double x;
    double sum1;
    double sum2;

    x = z[tid];
    if (x == 0) {
      hat_x = 0;
    }else{hat_x = log(abs(x));}

    if (x>0)
    {
        c1 = 10;
        c2 = 7.9;
        sign = 1;
    }
    else
    {
      c1 = 5.5;
      c2 = 3.1;
      if(x != 0)
      {
        sign  = -1 ;
      }else{sign = 0;}
    }
    x = sign * exp( hat_x + 0.049 * ( sin( c1 * hat_x ) + sin( c2* hat_x )  ) ) ;

    if(x > 0)
    {
      x = pow(x, 1 + beta * tid/((double) (dim-1)) * sqrt(x) );
    }
    x = x * pow(alpha, 0.5 * tid/((double) (dim-1)) );
    sum1 = x * x;
    sum2 = cos(2.0 * PI * x);
*/
    /*******************************************************************************
    //************Secod step is use reduction method calculate the sum**************
    ********************************************************************************/
    // First setp to save the sum1 and sum2 local data to the shard memory
    /*
    sharedSum1Array[threadIdx.x] = (tid < dim) ? sum1 : 0;
    sharedSum2Array[threadIdx.x] = (tid < dim) ? sum2 : 0;
    __syncthreads();
    //Step 2, reduction
    for (int s = blockDim.x/2; s > 0 ; s >>= 1) {
      if(threadIdx.x < s)
        sharedSum1Array[threadIdx.x] += sharedSum1Array[threadIdx.x + s];
        sharedSum2Array[threadIdx.x] += sharedSum2Array[threadIdx.x + s];
        __syncthreads();
    }
    //step 3, annd to the final output
    if(threadIdx.x == 0)
    {
      //atomicAdd((float *)d_sum1,(float)sharedSum1Array[0]);
      //atomicAdd((float *)d_sum2,(float)sharedSum2Array[0]);
      // for Cuda 8.0 it support double atomic add
      atomicAdd(d_sum1,sharedSum1Array[0]);
      atomicAdd(d_sum2,sharedSum2Array[0]);
    }


    */
}


double F3::compute(double*x){
  int    i;
  double result;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  if(Ovector == NULL) {
    // Ovector = createShiftVector(dimension,minX,maxX);
    Ovector = readOvector();
  }
  for(i = dimension - 1; i >= 0; i--) {
    anotherz[i] = x[i] - Ovector[i];
    //printf("%d\n",dimension );
  }
  hipEventRecord(start);
  result = ackley(anotherz,dimension);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("CPU running time (Only the calculation not include the inilization) %f ms\n", milliseconds);
  return(result);
}

//TODO GPU computing
//fp->GPUcompute(X,resultTest,run);
double F3::GPUcompute(double*x,double resultTest, unsigned run){
  int    i;
  double result = 0;
  double sum1 ;
  double sum2 ;

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  printf("\n*****************GPU Computing Result***************************\n\n");
  if(Ovector == NULL) {
    // Ovector = createShiftVector(dimension,minX,maxX);
    Ovector = readOvector();
  }
  for(i = dimension - 1; i >= 0; i--) {
    anotherz[i] = x[i] - Ovector[i];
  }
  // inilize the CUDA

  int threadsPerBlockDim = 512;
  int gridDimSize = (dimension + threadsPerBlockDim - 1) / threadsPerBlockDim;
  printf("GPU is running on Blocksize[%d],gridSize[%d]\n",threadsPerBlockDim,gridDimSize );
  // allocate the memory to device
  double* d_x , *d_sum1 , *d_sum2;
  hipMalloc(&d_x,dimension * sizeof(double));
  hipMalloc(&d_sum1,sizeof(double));
  hipMalloc(&d_sum2,sizeof(double));
  //Event start
  hipEventRecord(start);
  hipMemcpy(d_x, x, dimension * sizeof(float), hipMemcpyHostToDevice);
  hipMemset(d_sum1, 0.000 , sizeof(double));
  hipMemset(d_sum2, 0.000 , sizeof(double));

  testKernel<<<gridDimSize, threadsPerBlockDim>>>(d_x,dimension,d_sum1,d_sum2);
  //result = ackley(anotherz,dimension);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  //printf("Liang Xu\n" );
  //hipMemcpy(&sum1, d_sum1, sizeof(double), hipMemcpyDeviceToHost);
  //hipMemcpy(&sum2, d_sum2, sizeof(double), hipMemcpyDeviceToHost);
  hipError_t hipError_t = hipGetLastError();
    if(hipError_t != hipSuccess)
    {
        fprintf(stderr, "hipGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
        exit(EXIT_FAILURE);
    }

  result = sum1 + sum2;
  printf("Liang Xu\n" );
  if (abs(result - resultTest) > 0.1)
  {
    printf("Result in GPU is %f\n",result );
    printf("result not equal to the previous result in GPU computing\n" );
  }else{printf("GUP return the correct result\n");}
	hipEventElapsedTime(&milliseconds, start, stop);
	printf("GPU running time is %f ms\n", milliseconds);

  return(result);
}

// ackley function for single group non-separable
double Benchmarks::ackley(double*x,int dim){
  double sum1 = 0.0;
  double sum2 = 0.0;
  double sum;
  int    i;

  // T_{osz}
  transform_osz(x,dim);

  // T_{asy}^{0.2}
  transform_asy(x, 0.2, dim);

  // lambda
  Lambda(x, 10, dim);

  for(i = dim - 1; i >= 0; i--) {
    sum1 += (x[i] * x[i]);
    sum2 += cos(2.0 * PI * x[i]);
  }

  sum = -20.0 * exp(-0.2 * sqrt(sum1 / dim)) - exp(sum2 / dim) + 20.0 + E;
  return(sum);
}

// ackley function for m-group non-separable
double Benchmarks::ackley(double *x, int dim, int k)
{
  double sum1=0.0;
  double sum2=0.0;
  double result=0.0;
  int i;

  for(i=dim/k-1;i>=0;i--)
    {
      sum1+=x[Pvector[dim/k+i]]*x[Pvector[dim/k+i]];
      sum2+=cos(2.0*PI*x[Pvector[dim/k+i]]);
    }

  result=-20.0*exp(-0.2*sqrt(sum1/(dim/k)))-exp(sum2/(dim/k))+20.0+E;

  return(result);
}
