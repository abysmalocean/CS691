#include "hip/hip_runtime.h"
#include "F3.h"

/**
 * Shifted Ackley's Function
 *
 * as defined in "Benchmark Functions for the CEC'2010 Special Session
 * and Competition on Large-Scale Global Optimization" by Ke Tang,
 * Xiaodong Li, P. N. Suganthan, Zhenyu Yang, and Thomas Weise
 * published as technical report on January 8, 2010 at Nature Inspired
 * Computation and Applications Laboratory (NICAL), School of Computer
 * Science and Technology, University of Science and Technology of China,
 * Hefei, Anhui, China.
 */

F3::F3():Benchmarks(){
  Ovector = NULL;
  minX = -32;
  maxX = 32;
  ID = 3;
  anotherz = new double[dimension];

}

F3::~F3(){
  delete[] Ovector;
  delete[] anotherz;
}

double F3::compute(double*x){
  int    i;
  double result;
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;

  if(Ovector == NULL) {
    // Ovector = createShiftVector(dimension,minX,maxX);
    Ovector = readOvector();
  }
  for(i = dimension - 1; i >= 0; i--) {
    anotherz[i] = x[i] - Ovector[i];
    //printf("%d\n",dimension );
  }
  hipEventRecord(start);
  result = ackley(anotherz,dimension);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("CPU running time (Only the calculation not include the inilization) %f ms\n", milliseconds);
  return(result);
}


// ackley function for single group non-separable
double Benchmarks::ackley(double*x,int dim){
  double sum1 = 0.0;
  double sum2 = 0.0;
  double sum;
  int    i;

  // T_{osz}
  transform_osz(x,dim);

  // T_{asy}^{0.2}
  transform_asy(x, 0.2, dim);

  // lambda
  Lambda(x, 10, dim);

  for(i = dim - 1; i >= 0; i--) {
    sum1 += (x[i] * x[i]);
    sum2 += cos(2.0 * PI * x[i]);
  }

  sum = -20.0 * exp(-0.2 * sqrt(sum1 / dim)) - exp(sum2 / dim) + 20.0 + E;
  return(sum);
}

// ackley function for m-group non-separable
double Benchmarks::ackley(double *x, int dim, int k)
{
  double sum1=0.0;
  double sum2=0.0;
  double result=0.0;
  int i;

  for(i=dim/k-1;i>=0;i--)
    {
      sum1+=x[Pvector[dim/k+i]]*x[Pvector[dim/k+i]];
      sum2+=cos(2.0*PI*x[Pvector[dim/k+i]]);
    }

  result=-20.0*exp(-0.2*sqrt(sum1/(dim/k)))-exp(sum2/(dim/k))+20.0+E;

  return(result);
}
