
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(float *A, float *B, float *C, int numElements)
{
	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid < numElements)
		C[tid] = A[tid] + B[tid];
}

int main(int argc, char* argv[])
{
	int numElements = pow(2,20); // 2^20 approximately 1M elements

	// Allocate host memory
	float *h_A, *h_B, *h_C;

	hipHostMalloc(&h_A, numElements * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&h_B, numElements * sizeof(float), hipHostMallocDefault);
	hipHostMalloc(&h_C, numElements * sizeof(float), hipHostMallocDefault);

	// Initialize the host input vectors
	for (int i = 0; i < numElements; ++i)
	{
		h_A[i] = rand()/(float)RAND_MAX;
		h_B[i] = rand()/(float)RAND_MAX;
	}

	float *d_A1, *d_A2, *d_B1, *d_B2, *d_C1, *d_C2;
	hipEvent_t start1, stop1, start2, stop2;
	hipStream_t stream1, stream2;

	float milliseconds1, milliseconds2;

	int threadsPerBlock = 256;
	int blocksPerGrid = (numElements/2 + threadsPerBlock - 1) / threadsPerBlock;

	hipSetDevice(0);

	hipMalloc(&d_A1, numElements/2 * sizeof(float));
	hipMalloc(&d_B1, numElements/2 * sizeof(float));
	hipMalloc(&d_C1, numElements/2 * sizeof(float));

	hipStreamCreate(&stream1);
	hipEventCreate(&start1);
	hipEventCreate(&stop1);

	hipSetDevice(1);

	hipMalloc(&d_A2, numElements/2 * sizeof(float));
	hipMalloc(&d_B2, numElements/2 * sizeof(float));
	hipMalloc(&d_C2, numElements/2 * sizeof(float));

	hipStreamCreate(&stream2);
	hipEventCreate(&start2);
	hipEventCreate(&stop2);

	hipSetDevice(0);

	hipEventRecord(start1, stream1);

	hipMemcpyAsync(d_A1, h_A, numElements/2 * sizeof(float), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_B1, h_B, numElements/2 * sizeof(float), hipMemcpyHostToDevice, stream1);

	vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream1>>>(d_A1, d_B1, d_C1, numElements/2);

	hipMemcpyAsync(h_C, d_C1, numElements/2 * sizeof(float), hipMemcpyDeviceToHost, stream1);

	hipEventRecord(stop1, stream1);

	hipSetDevice(1);

	hipEventRecord(start2, stream2);

	hipMemcpyAsync(d_A2, &h_A[numElements/2], numElements/2 * sizeof(float), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(d_B2, &h_B[numElements/2], numElements/2 * sizeof(float), hipMemcpyHostToDevice, stream2);

	vectorAdd<<<blocksPerGrid, threadsPerBlock, 0, stream2>>>(d_A2, d_B2, d_C2, numElements/2);

	hipMemcpyAsync(&h_C[numElements/2], d_C2, numElements/2 * sizeof(float), hipMemcpyDeviceToHost, stream2);

	hipEventRecord(stop2, stream2);

	hipSetDevice(0);
	hipEventSynchronize(stop1);
	//cudaStreamSynchronize(stream1);
	hipSetDevice(1);
	hipEventSynchronize(stop2);
	//cudaStreamSynchronize(stream2);

	hipEventElapsedTime(&milliseconds1, start1, stop1);
	hipEventElapsedTime(&milliseconds2, start2, stop2);

	printf("GPU %d time %f ms\n", 0, milliseconds1);
	printf("GPU %d time %f ms\n", 1, milliseconds2);

	hipError_t hipError_t = hipGetLastError();

	if(hipError_t != hipSuccess)
	{
		fprintf(stderr, "cudaGetLastError() returned %d: %s\n", hipError_t, hipGetErrorString(hipError_t));
		exit(EXIT_FAILURE);
	}

	// Verify that the result vector is correct
	for (int i = 0; i < numElements; i++)
		if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
		{
			fprintf(stderr, "Result verification failed at element %d!\n", i);
			exit(EXIT_FAILURE);
		}

	printf("Sum of the vectors was OK\n");

	hipSetDevice(0);
	hipFree(d_A1);
	hipFree(d_B1);
	hipFree(d_C1);

	hipSetDevice(1);
	hipFree(d_A2);
	hipFree(d_B2);
	hipFree(d_C2);

	// Free host memory
	hipHostFree(h_A);
	hipHostFree(h_B);
	hipHostFree(h_C);

	return 0;
}

